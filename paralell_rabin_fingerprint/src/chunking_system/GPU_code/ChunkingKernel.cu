#include "hip/hip_runtime.h"
/**
 * ChunkingKernel.cu
 *
 *  Created on: Jul 2, 2013
 *      Author: Zahari Dichev <zaharidichev@gmail.com>
 */

#ifndef CHUNKINGKERNEL_CU_
#define CHUNKINGKERNEL_CU_

#include "../../rabin_fingerprint/Chunker.h"
#include  "hip/hip_runtime.h"
#include "ResourceManagement.h"
#include "KernelStarter.h"
#include "BitFieldArray.h"
#include <iostream>
#include <fstream>      // std::ifstream
#include "../../etc/helpers/Macros.h"
#include "hashing/sha1_kernel.cu"
#include "openssl/sha.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

__device__ int getThrID() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ void getThreadBounds(threadBounds* bounds, int dataLn, int threadsUsed, int thrID, int workPerThr) {

	bounds->start = thrID * workPerThr;

	//ACCOUTN FOR ANY LEFTOVER DATA THAT CANNOT BE DISTRIBUTED ;)
	bounds->end = (thrID == threadsUsed - 1) ? bounds->end = dataLn : bounds->start + workPerThr;

}

__global__ void findBreakPointsSegmented(rabinData* deviceRabin, chunkingContext* ctx, BYTE* data, int dataLen, int* results, int threadsUsed, BYTE* hashes) {

	int thrID = getThrID();

	if (thrID < threadsUsed) {

		threadBounds dataBounds;
		getThreadBounds(&dataBounds, dataLen, threadsUsed, thrID, ctx->workPerThread);

		chunkDataWithLimits(deviceRabin, data, dataBounds, ctx, results, threadsUsed, hashes);
		//chunkDataWithLimits(deviceRabin, data, dataBounds, ctx, results);

	}

}

__global__ void findBreakPointsFreeMode(rabinData* deviceRabin, BYTE* data, int dataLen, bitFieldArray results, int threadsUsed, int workPerThread,
		int divisor) {

	int thrID = getThrID();

	if (thrID < threadsUsed) {

		threadBounds dataBounds;

		getThreadBounds(&dataBounds, dataLen, threadsUsed, thrID, workPerThread);

		chunkDataFreeMode(deviceRabin, data, dataBounds, divisor, results, threadsUsed);
	}
}

void startCreateBreakpointsKernel(int blocksSize, int numBlocks, rabinData* deviceRabin, BYTE* deviceData, int dataLen, bitFieldArray results, int threadsUsed,
		int workPerThread, int D) {
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 5242880);

	findBreakPointsFreeMode<<<numBlocks, blocksSize>>>(deviceRabin, deviceData, dataLen, results, threadsUsed, workPerThread, D);

	gpuErrchk(hipGetLastError());

	hipDeviceSynchronize();
}

void startSegmentedChunkingAndHashingKernel(size_t blocksSize, size_t numBlocks, rabinData* rabinData_d, chunkingContext* ctx_d, BYTE* dataToChunk_d,
		size_t sizeOfData, size_t activeThreads, BYTE* hashed_d, int* results_d) {


	hipDeviceSetLimit(hipLimitPrintfFifoSize, 5242880);

	findBreakPointsSegmented<<<numBlocks, blocksSize>>>(rabinData_d,ctx_d,dataToChunk_d,sizeOfData,results_d,activeThreads,hashed_d);

	gpuErrchk(hipGetLastError());

	hipDeviceSynchronize();
}

int __host__ getSizeOfBPArray(int dataLn, int minThreshold) {
	return (dataLn % minThreshold == 0) ? dataLn / minThreshold : (dataLn / minThreshold) + 1;
}

int f() {

	std::ifstream infile("/home/zahari/Desktop/data.txt", std::ofstream::binary);

	int sizeOfData = 536870912;
	int minSize = 32768;
	int maxSize = 131072;

	int sizeOfBParray = getSizeOfBPArray(sizeOfData, minSize);
	unsigned char* data = (unsigned char*) malloc(sizeOfData);
	infile.read((char*) data, sizeOfData);

	// host and device data for rabin window context
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 5242880);
	rabinData hostData;
	rabinData* deviceData;

	// we first init the window on the host.
	initWindow(&hostData, 0xbfe6b8a5bf378d83);

	//allocate device memory for the rabin context data

	CUDA_CHECK_RETURN(hipMalloc((void** ) &deviceData, sizeof(rabinData)));
	//copy the data to device
	CUDA_CHECK_RETURN(hipMemcpy(deviceData, &hostData, sizeof(rabinData), hipMemcpyHostToDevice));

	// allocate space for the data that we need to chunk and copy it to the device
	BYTE* dataToFingerprint_d;
	CUDA_CHECK_RETURN(hipMalloc((void** ) &dataToFingerprint_d, sizeof(BYTE) * sizeOfData));
	CUDA_CHECK_RETURN(hipMemcpy(dataToFingerprint_d, data, sizeof(BYTE) * sizeOfData, hipMemcpyHostToDevice));

	//now we allocate some space for the results
	int* resultingBreakpoints_d;
	CUDA_CHECK_RETURN(hipMalloc((void** ) &resultingBreakpoints_d, sizeof(int) * sizeOfBParray));
	CUDA_CHECK_RETURN(hipMemset(resultingBreakpoints_d, 0, sizeof(int) * sizeOfBParray));

	int threadsNeeded = getNumNeededThreads(sizeOfData, 262144);

	int blocksize = 160;

	int numBlocks = threadsNeeded / blocksize;
	if (threadsNeeded % blocksize) {
		++numBlocks;
	}

	BYTE* hashes = (BYTE*) (malloc(sizeof(BYTE) * sizeOfBParray * 20));

	BYTE* hashes_d;
	CUDA_CHECK_RETURN(hipMalloc((void** ) &hashes_d, sizeof(BYTE) * sizeOfBParray * 20));

	int bpsPerThread = round(((double) (sizeOfBParray)) / threadsNeeded);

	chunkingContext ctx;

	ctx.BpreakpointsPerThread = bpsPerThread;
	ctx.D = 512;
	ctx.Ddash = 256;
	ctx.maxThr = maxSize;
	ctx.minThr = minSize;
	ctx.sizeOfBreakpointsArray = sizeOfBParray;
	ctx.workPerThread = 262144;

	chunkingContext* ctx_d;

	CUDA_CHECK_RETURN(hipMalloc((void** ) &ctx_d, sizeof(chunkingContext)));
	CUDA_CHECK_RETURN(hipMemcpy(ctx_d, &ctx, sizeof(chunkingContext), hipMemcpyHostToDevice));

	findBreakPointsSegmented<<<numBlocks, blocksize>>>(deviceData, ctx_d, dataToFingerprint_d, sizeOfData, resultingBreakpoints_d, threadsNeeded, hashes_d);

	int* resultingBreakpoints = (int*) malloc(sizeof(int) * sizeOfBParray);

	//copy back into our supplied data
	CUDA_CHECK_RETURN(hipMemcpy(resultingBreakpoints, resultingBreakpoints_d, sizeof(int) * sizeOfBParray, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(hashes, hashes_d, sizeof(BYTE) * sizeOfBParray * 20, hipMemcpyDeviceToHost));

	// free all the memory alocated on the cardgetThreadBounds
	CUDA_CHECK_RETURN(hipFree(deviceData));
	CUDA_CHECK_RETURN(hipFree(dataToFingerprint_d));
	CUDA_CHECK_RETURN(hipFree(resultingBreakpoints_d));

	for (int var = 0; var < sizeOfBParray; ++var) {

		printf("%d\n", resultingBreakpoints[var]);

	}

	for (int i = 0; i < sizeOfBParray; ++i) {

		for (int var = 0; var < 20; ++var) {
			printf("%02x", hashes[i * 20 + var]);

		}
		printf("\n");

	}

	free(data);
	free(resultingBreakpoints);

	unsigned char* buffer = (unsigned char*) malloc(33135);
	std::ifstream infile2("/home/zahari/Desktop/data.txt", std::ofstream::binary);
	infile.seekg(536775699);

	infile.read((char*) buffer, 33135);
	BYTE* digest = (BYTE*) malloc(20);

	SHA1(buffer, 33135, digest);
	printf("----------------------------------\n");

	for (int var = 0; var < 20; ++var) {
		printf("%02x", digest[var]);

	}

}

#endif /* CHUNKINGKERNEL_CU_ */
